#include "hip/hip_runtime.h"
// MP 1
#include	<wb.h>

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    //@@ Insert code to implement vector addition here
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < len) out[i] = in1[i] + in2[i];
}

//__host__ void checkCudaError(hipError_t err, int line) {
//  if (err != hipSuccess) {
//     printf(\"%s in %s at line %d\\n\", cutaGetErrorString(err), __FILE__, line);
//     //exit(EXIT_FAILURE);
//  }
//}

int main(int argc, char ** argv) {
    wbArg_t args;
    int inputLength;
    float * hostInput1;
    float * hostInput2;
    float * hostOutput;
    float * deviceInput1;
    float * deviceInput2;
    float * deviceOutput;
      
    args = wbArg_read(argc, argv);

    wbTime_start(Generic, \"Importing data and creating memory on host\");
    hostInput1 = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *) malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, \"Importing data and creating memory on host\");

    wbLog(TRACE, \"The input length is \", inputLength);
    
    int size = inputLength * sizeof(float);
  
	wbTime_start(GPU, \"Allocating GPU memory.\");
    //@@ Allocate GPU memory here
//    checkCudaError(hipMalloc((void **) &deviceInput1, size), __LINE__);
//    checkCudaError(hipMalloc((void **) &deviceInput2, size), __LINE__);
//    checkCudaError(hipMalloc((void **) &deviceOutput, size), __LINE__);
    hipMalloc((void **) &deviceInput1, size);
    hipMalloc((void **) &deviceInput2, size);
    hipMalloc((void **) &deviceOutput, size);
  
    wbTime_stop(GPU, \"Allocating GPU memory.\");

    wbTime_start(GPU, \"Copying input memory to the GPU.\");
    //@@ Copy memory to the GPU here

    hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
    hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
  
    wbTime_stop(GPU, \"Copying input memory to the GPU.\");
    
    //@@ Initialize the grid and block dimensions here
    dim3 DimGrid ((inputLength-1)/256 + 1, 1, 1);
    dim3 DimBlock(256, 1, 1);

    wbTime_start(Compute, \"Performing CUDA computation\");
    //@@ Launch the GPU Kernel here
    vecAdd<<<DimGrid,DimBlock>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);
      
    hipDeviceSynchronize();
    wbTime_stop(Compute, \"Performing CUDA computation\");
    
    wbTime_start(Copy, \"Copying output memory to the CPU\");
    //@@ Copy the GPU memory back to the CPU here
    hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);
      
    wbTime_stop(Copy, \"Copying output memory to the CPU\");

    wbTime_start(GPU, \"Freeing GPU Memory\");
    //@@ Free the GPU memory here
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);

    wbTime_stop(GPU, \"Freeing GPU Memory\");

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}